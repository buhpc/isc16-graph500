#include "hip/hip_runtime.h"
/**
 * Implementing Breadth first search on CUDA using algorithm given in HiPC'07
 * paper "Accelerating Large Graph Algorithms on the GPU using CUDA"
 *
 * Copyright (c) 2008 
 * International Institute of Information Technology - Hyderabad. 
 * All rights reserved.
 *
 * Permission to use, copy, modify and distribute this software and its 
 * documentation for educational purpose is hereby granted without fee, 
 * provided that the above copyright notice and this permission notice 
 * appear in all copies of this software and that you do not sell the software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, 
 * IMPLIED OR OTHERWISE.
 *
 * Created by Pawan Harish.
 *
 * Modified by Boston Green Team.
 */

__global__ void Kernel2( bool* g_graph_mask, bool *g_updating_graph_mask, bool* g_graph_visited, bool *g_over ) {
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if (tid < VERTICES && g_updating_graph_mask[tid]) {
		g_graph_mask[tid] = true;
		g_graph_visited[tid] = true;
		*g_over = true;
		// cuPrintf("Visiting: branches in kernel2, %d,", tid);
		g_updating_graph_mask[tid] = false;
	}
}