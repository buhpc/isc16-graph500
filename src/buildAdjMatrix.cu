#include "hip/hip_runtime.h"
/**
 * filename: buildAdjMatirx.cu
 * contents: contains the cuda kernel to build the adjancecy
 * matrix from the list of edges
 */

#include "buildAdjMatrix.h"

#include <stdio.h>

/**
 * wrapper for cuda kernel
 */
void buildGraph(int threadsPerBlock,
                int numBlocks,
                int *adjMatrix,
                int numNodes,
                long long *edgeList,
                int numEdges,
                int offset,
                int graphSize,
                int rank) {
  // launch kernel
  buildAdjMatrix<<<numBlocks, threadsPerBlock>>>(adjMatrix,
                                                 numNodes,
                                                 edgeList,
                                                 numEdges,
                                                 offset,
                                                 graphSize, 
                                                 rank);
}

/**
 * constructs and adjacency matrix in device memory from the list
 * of edges
 */
__global__ void buildAdjMatrix(int *adjMatrix,
                               int numNodes,
                               long long *edgeList,
                               int numEdges,
                               int offset,
                               int graphSize,
                               int rank) {

  // each thread gets 1 edges in the edge list to build
  int index = blockIdx.x*blockDim.x + threadIdx.x;


  if (index < numEdges) {
    if (rank == 1) {
      printf("(rank, index) = (%d, %d)\n", rank, index);
    } 
    // get the two vertices to connect
    int vertA = edgeList[index];
    int vertB = edgeList[index + numEdges];
    
    // remove self edges
    if (vertA == vertB) { return; }



    // set edge in both direction
    if (vertA >= offset && vertA < (offset + graphSize)) {
      // vertA is row vertB is column
      atomicOr(&adjMatrix[vertA*numNodes+vertB], 1);
    }
    if (vertB >= offset && vertB < offset + graphSize) {
      // vert b is row vertA is column
      atomicOr(&adjMatrix[vertB*numNodes+vertA], 1);
    }
  }

}
